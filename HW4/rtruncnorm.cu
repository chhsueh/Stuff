#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{

__global__ void 
rtruncnorm_kernel(
    float *x, 
    int n, 
    float *mu, 
    float *sigma, 
    float *lo, 
    float *hi,
    int maxtries,
    int rngnum) //number of the random seed
{
    int accepted;
    float sample;
    int numtries;
    float m;
    float alpha;

    //variables for rejection sampling
    float rexp; 
    float z; 
    float phi;
    float u;

    // Usual block/thread indexing...
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    
    // map these into a single index : idx
    int idx = myblock * blocksize + subthread;

    // check whether the idx is <n
    if (idx < n)
    {

        // Setup the RNG:
        hiprandState rng_state;
        hiprand_init(9131+idx*17,rngnum,0,&rng_state);

        // Sample:
        accepted = 0;
        numtries = 0;

        while (accepted == 0 && numtries < maxtries)
        {
            sample = mu[idx]+sigma[idx]*hiprand_normal(&rng_state);
            numtries = numtries+1;

            if (sample>lo[idx] && sample<hi[idx])
            {

                accepted = 1;
                x[idx] = sample;
                //printf("rnumber = %f\n", sample);

            } //end of if(small) loop

        } // end of while loop

	while (accepted == 0) //if accepted = 0 run rejection sampling.
	{ 
	    //code for rejection sampling.
	    if(abs(lo[idx]-mu[idx]) < abs(hi[idx]-mu[idx])){ //right tail

		m = abs((lo[idx]-mu[idx])/sigma[idx]);
		alpha = (m+sqrt(pow(m,2)+4))/2;
	    	rexp = -log(hiprand_uniform(&rng_state))/alpha;
	    	z = m + rexp;
		if (m<alpha){
			phi = exp(-pow(alpha-z,2)/2);
	    	}
	    	else{
			phi = exp(pow(m-alpha,2)/2-pow(alpha-z,2)/2);
	    	} //decide phi

		u = hiprand_uniform(&rng_state);
		if (u<phi){
			accepted = 1;
			x[idx] = mu[idx]+sigma[idx]*z;
	    	} 
		
	    } else{ //left tail

		m = abs((mu[idx]-hi[idx])/sigma[idx]);
		alpha = (m+sqrt(pow(m,2)+4))/2;
	    	rexp = -log(hiprand_uniform(&rng_state))/alpha;
	    	z = m + rexp;
		if (m<alpha){
		phi = exp(-pow(alpha-z,2)/2);
	    	}
	    	else{
			phi = exp(pow(m-alpha,2)/2-pow(alpha-z,2)/2);
	    	} //decide phi

		u = hiprand_uniform(&rng_state);
		if (u<phi){
			accepted = 1;
			x[idx] = mu[idx]-sigma[idx]*z;
	    	} 

	    }
	
		
	} // end of rejection sampling.

    } // end of if loop

    return;
} // end of function

} // END extern "C"


//#### More variables: ########################
                  //int mu_len, 
		  //int sigma_len,
                  //int lo_len, 
		  //int hi_len,
                  //int maxtries
//#############################################
